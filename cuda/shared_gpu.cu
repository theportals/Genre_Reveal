#include "hip/hip_runtime.h"
//
// Created by Tom on 11/29/2023.
//
#include <iostream>
#include "../point.hpp"

string xcol;
string ycol;
string zcol;

double converge_threshold = 1e-7;

__global__ void updateCentroids(Point* points, Point* centroids, int* nPoints, double* sumX, double* sumY, double* sumZ, int k, int n);

int main(int argc, char* argv[]) {
    string filepath;

    if (argc != 7) {
        cout << "Usage: shared_gpu.exe <CUDA device id> <filepath to csv> <number of clusters> <x column key> <y column key> <z column key>" << endl;
        return -1;
    }

    // Initialize arguments
    int device = strtol(argv[1], nullptr, 10);
    filepath = argv[2];
    int k = strtol(argv[3], nullptr, 10);
    xcol = argv[4];
    ycol = argv[5];
    zcol = argv[6];
    hipDeviceProp_t properties{};
    hipGetDeviceProperties(&properties, device);
    int blockSize = properties.maxThreadsPerBlock;

    // Read from csv file
    auto before = chrono::high_resolution_clock::now();
    cout << "Loading points from csv (this may take a while)..." << endl;
    vector<Point> points = readcsv(filepath, xcol, ycol, zcol);
    auto after = chrono::high_resolution_clock::now();
    auto duration = chrono::duration_cast<chrono::milliseconds>(after - before);
    cout << points.size() << " points loaded in " << duration.count() << "ms." << endl;

    // Pick k points at random to create centroids
    vector<Point> centroids;
    srand(123);
    for (int i = 0; i < k; i++) {
        centroids.push_back(points[rand() % points.size()]);
    }

    // Used for calculating averages of cluster locations
    auto nPoints_h = new int[k];
    fill(&nPoints_h[0], &nPoints_h[k], 0);
    auto nPoints_d = new int[k];
    auto sumX_h = new double[k];
    auto sumY_h = new double[k];
    auto sumZ_h = new double[k];
    fill(&sumX_h[0], &sumX_h[k], 0.0);
    fill(&sumY_h[0], &sumY_h[k], 0.0);
    fill(&sumZ_h[0], &sumZ_h[k], 0.0);
    auto sumX_d = new double[k];
    auto sumY_d = new double[k];
    auto sumZ_d = new double[k];

    // Allocate device memory
    hipError_t cudaErr = hipSuccess;
    // CUDA likes arrays more than vectors
    Point* points_h = points.data();
    Point* centroids_h = centroids.data();
    Point* points_d;
    Point* centroids_d;
    cudaErr = hipMalloc((void **) &points_d, sizeof(Point) * points.size());
    cudaErr = hipMalloc((void **) &centroids_d, sizeof(Point) * k);
    cudaErr = hipMalloc((void **) &nPoints_d, sizeof(int) * k);
    cudaErr = hipMalloc((void **) &sumX_d, sizeof(double) * k);
    cudaErr = hipMalloc((void **) &sumY_d, sizeof(double) * k);
    cudaErr = hipMalloc((void **) &sumZ_d, sizeof(double) * k);
    hipDeviceSynchronize();

    if (cudaErr != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory (error code %s)\n", hipGetErrorString(cudaErr));
        exit(-2);
    }

    // Copy host variables to device
    cudaErr = hipMemcpy(points_d, points_h, sizeof(Point) * points.size(), hipMemcpyHostToDevice);
    cudaErr = hipMemcpy(centroids_d, centroids_h, sizeof(Point) * k, hipMemcpyHostToDevice);
    cudaErr = hipMemcpy(nPoints_d, nPoints_h, sizeof(int) * k, hipMemcpyHostToDevice);
    cudaErr = hipMemcpy(sumX_d, sumX_h, sizeof(double) * k, hipMemcpyHostToDevice);
    cudaErr = hipMemcpy(sumY_d, sumY_h, sizeof(double) * k, hipMemcpyHostToDevice);
    cudaErr = hipMemcpy(sumZ_d, sumZ_h, sizeof(double) * k, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    if (cudaErr != hipSuccess) {
        fprintf(stderr, "Failed to copy host variables to device (error code %s)\n", hipGetErrorString(cudaErr));
        exit(-3);
    }

    // Update step
    int epochs = 0;
    bool hasConverged = false;
    before = chrono::high_resolution_clock::now();
    while (!hasConverged) {
        epochs++;

        // Assign each point to the nearest centroid
        updateCentroids<<<ceil((double) points.size() / blockSize), blockSize>>>(points_d, centroids_d, nPoints_d, sumX_d, sumY_d, sumZ_d, k, points.size());
        cudaErr = hipDeviceSynchronize();

        if (cudaErr != hipSuccess) {
            fprintf(stderr, "Failed to start kernel (error code %s)\n", hipGetErrorString(cudaErr));
            exit(-4);
        }

        // Retrieve updated sums from device
        // TODO: Make a function so we don't have this code in like twelve different 4-line blocks
        cudaErr = hipMemcpy(nPoints_h, nPoints_d, sizeof(int) * k, hipMemcpyDeviceToHost);
        cudaErr = hipMemcpy(sumX_h, sumX_d, sizeof(double) * k, hipMemcpyDeviceToHost);
        cudaErr = hipMemcpy(sumY_h, sumY_d, sizeof(double) * k, hipMemcpyDeviceToHost);
        cudaErr = hipMemcpy(sumZ_h, sumZ_d, sizeof(double) * k, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        if (cudaErr != hipSuccess) {
            fprintf(stderr, "Failed to copy updated points from device (error code %s)\n", hipGetErrorString(cudaErr));
            exit(-5);
        }


        // Compute the new centroids. Because we usually have a small k, this doesn't need to be parallel
        bool shouldEnd = true;
        for (int j = 0; j < k; j++) {
            double oldx = centroids_h[j].x;
            double oldy = centroids_h[j].y;
            double oldz = centroids_h[j].z;

            centroids_h[j].x = sumX_h[j] / nPoints_h[j];
            centroids_h[j].y = sumY_h[j] / nPoints_h[j];
            centroids_h[j].z = sumZ_h[j] / nPoints_h[j];

            double distMoved = (centroids[j].x - oldx) * (centroids[j].x - oldx) +
                               (centroids[j].y - oldy) * (centroids[j].y - oldy) +
                               (centroids[j].z - oldz) * (centroids[j].z - oldz);

            if (distMoved > converge_threshold)
                shouldEnd = false;
        }

        hasConverged = shouldEnd;
        if (!hasConverged) {
            // If we haven't converged, copy new centroids to device, reset sums and nPoints
            for (int j = 0; j < k; j++) {
                nPoints_h[j] = 0;
                sumX_h[j] = 0.0;
                sumY_h[j] = 0.0;
                sumZ_h[j] = 0.0;
            }
            cudaErr = hipMemcpy(nPoints_d, nPoints_h, sizeof(int) * k, hipMemcpyHostToDevice);
            cudaErr = hipMemcpy(sumX_d, sumX_h, sizeof(double) * k, hipMemcpyHostToDevice);
            cudaErr = hipMemcpy(sumY_d, sumY_h, sizeof(double) * k, hipMemcpyHostToDevice);
            cudaErr = hipMemcpy(sumZ_d, sumZ_h, sizeof(double) * k, hipMemcpyHostToDevice);
            cudaErr = hipMemcpy(centroids_d, centroids_h, sizeof(Point) * k, hipMemcpyHostToDevice);
            hipDeviceSynchronize();

            if (cudaErr != hipSuccess) {
                fprintf(stderr, "Failed to copy new centroids to device (error code %s)\n", hipGetErrorString(cudaErr));
                exit(-6);
            }
        }
    }
    after = chrono::high_resolution_clock::now();
    duration = chrono::duration_cast<chrono::milliseconds>(after - before);
    cout << "Clustered with " << epochs << " epochs in " << duration.count() << "ms." << endl;

    // Copy converged points from device
    cudaErr = hipMemcpy(points_h, points_d, sizeof(Point) * points.size(), hipMemcpyDeviceToHost);

    if (cudaErr != hipSuccess) {
        fprintf(stderr, "Failed to copy converged points from device (error code %s)\n", hipGetErrorString(cudaErr));
        exit(-7);
    }

    // Write to file
    ofstream myfile;
    myfile.open("output.csv");
    myfile << "x,y,z,c" << endl;
    for (int i = 0; i < points.size(); i++) {
        Point point = points_h[i];
        myfile << point.x << "," << point.y << "," << point.z << "," << point.cluster << endl;
    }
    myfile.close();
    cout << "Written to output.csv" << endl;
}

__global__ void updateCentroids(Point* points, Point* centroids, int* nPoints, double* sumX, double* sumY, double* sumZ, int k, int n) {
    // Calculate global thread index
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int bx = blockDim.x;

    int gid = bx*bid + tid;
    if (gid < n) {
        Point p = points[gid];
        for (int j = 0; j < k; j++) {
            Point c = centroids[j];
            double dist = (p.x - c.x) * (p.x - c.x) + (p.y - c.y) * (p.y - c.y) + (p.z - c.z) * (p.z - c.z);
            if (dist < p.minDist) {
                p.minDist = dist;
                p.cluster = j;
            }
        }

        // Rather than making a second kernel function, append data to centroids here
        int cluster = p.cluster;
        atomicAdd(&nPoints[cluster], 1);
        atomicAdd_block(&sumX[cluster], p.x);
        atomicAdd_block(&sumY[cluster], p.y);
        atomicAdd_block(&sumZ[cluster], p.z);

        p.minDist = DBL_MAX; // reset distance
        points[gid] = p;
    }
}