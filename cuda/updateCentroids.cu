#include "hip/hip_runtime.h"
/*
    Created by Bridger 12/4/2023
    Modeled after shared_gpu
    Compilation: nvcc -c updateCentroids.cu -o updateCentroids.o
*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../point.h"

inline hipError_t checkCuda(hipError_t result, const string& errorMessage) {
    if (result != hipSuccess) {
        fprintf(stderr, "%s\n", errorMessage.c_str());
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        exit(-1);
    }
    return result;
}

// Function to perform atomic add on a double
__device__ double atomicAddDouble(double* address, double val) {
    auto* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old_val, new_val;
    do {
        old_val = *address_as_ull;
        new_val = __double_as_longlong(val + __longlong_as_double(old_val));
    } while (atomicCAS(address_as_ull, old_val, new_val) != old_val);
    return __longlong_as_double(old_val);
}

__global__ void updateCentroids(Point* points, Point* centroids, int* nPoints, double* sumX, double* sumY, double* sumZ, int k, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < n) {
        // Assign each point to its nearest centroid
        Point p = points[tid];

        for (int j = 0; j < k; j++) {
            Point c = centroids[j];
            double dist = (p.x - c.x) * (p.x - c.x) + (p.y - c.y) * (p.y - c.y) + (p.z - c.z) * (p.z - c.z);

            if (dist < p.minDist) {
                p.minDist = dist;
                p.cluster = j;
            }
        }

        // Append data to centroids
        int cluster = p.cluster;
        atomicAdd(&nPoints[cluster], 1);
        atomicAddDouble(&sumX[cluster], p.x);
        atomicAddDouble(&sumY[cluster], p.y);
        atomicAddDouble(&sumZ[cluster], p.z);

        p.minDist = DBL_MAX; // reset distance
        points[tid] = p;
    }
}

extern "C" void launch_update_centroids(Point* points, Point* centroids, int* nPoints, double* sumX, double* sumY, double* sumZ, int k, int n, int thread) {
    //Dimensions
    int blockSize = 128;
    int numBlocks = ceil((double) n / blockSize);

    // GPU memory allocation
    Point* points_d;
    Point* centroids_d;
    int* nPoints_d;
    double* sumX_d;
    double* sumY_d;
    double* sumZ_d;
    checkCuda(hipMalloc((void **)&points_d, sizeof(Point) * n), "Could not allocate points.");
    checkCuda(hipMalloc((void **)&centroids_d, sizeof(Point) * k), "Could not allocate centroids.");
    checkCuda(hipMalloc((void **)&nPoints_d, sizeof(int) * k), "Could not allocate nPoints.");
    checkCuda(hipMalloc((void **)&sumX_d, sizeof(double) * k), "Could not allocate sumX.");
    checkCuda(hipMalloc((void **)&sumY_d, sizeof(double) * k), "Could not allocate sumY.");
    checkCuda(hipMalloc((void **)&sumZ_d, sizeof(double) * k), "Could not allocate sumZ.");

    // Copy data from CPU to GPU
    checkCuda(hipMemcpy(points_d, points, sizeof(Point) * n, hipMemcpyHostToDevice), "Could not copy points.");
    checkCuda(hipMemcpy(centroids_d, centroids, sizeof(Point) * k, hipMemcpyHostToDevice), "Could not copy centroids.");
    checkCuda(hipMemcpy(nPoints_d, nPoints, sizeof(int) * k, hipMemcpyHostToDevice), "Could not copy nPoints.");
    checkCuda(hipMemcpy(sumX_d, sumX, sizeof(double) * k, hipMemcpyHostToDevice), "Could not copy sumX.");
    checkCuda(hipMemcpy(sumY_d, sumY, sizeof(double) * k, hipMemcpyHostToDevice), "Could not copy sumY.");
    checkCuda(hipMemcpy(sumZ_d, sumZ, sizeof(double) * k, hipMemcpyHostToDevice), "Could not copy sumZ.");

    // Run kernel
    updateCentroids<<<numBlocks, blockSize>>> (points_d, centroids_d, nPoints_d, sumX_d, sumY_d, sumZ_d, k, n);

    // Wait for the kernel to finish
    string message;
    message.append("Thread ").append(to_string(thread)).append(" could not run kernel.");
    checkCuda(hipDeviceSynchronize(), message);

    // Copy the result back from GPU to CPU
    checkCuda(hipMemcpy(points, points_d, sizeof(Point) * n, hipMemcpyDeviceToHost), "Could not copy points from device.");
    checkCuda(hipMemcpy(centroids, centroids_d, sizeof(Point) * k, hipMemcpyDeviceToHost), "Could not copy centroids from device.");
    checkCuda(hipMemcpy(nPoints, nPoints_d, sizeof(int) * k, hipMemcpyDeviceToHost), "Could not copy nPoints from device.");
    checkCuda(hipMemcpy(sumX, sumX_d, sizeof(double) * k, hipMemcpyDeviceToHost), "Could not copy sumX from device.");
    checkCuda(hipMemcpy(sumY, sumY_d, sizeof(double) * k, hipMemcpyDeviceToHost), "Could not copy sumY from device.");
    checkCuda(hipMemcpy(sumZ, sumZ_d, sizeof(double) * k, hipMemcpyDeviceToHost), "Could not copy sumZ from device.");

    // Free GPU memory
    hipFree(points_d);
    hipFree(centroids_d);
    hipFree(nPoints_d);
    hipFree(sumX_d);
    hipFree(sumY_d);
    hipFree(sumZ_d);

}


