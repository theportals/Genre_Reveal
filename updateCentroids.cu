#include "hip/hip_runtime.h"
/*
    Created by Bridger 12/4/2023
    Modeled after shared_gpu
    Compilation: nvcc -c multiply.cu -o multiply.exe
*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "point.h"

__global__ void updateCentroids(Point* points, Point* centroids, int* nPoints, double* sumX, double* sumY, double* sumZ, int k, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < n) {
        Point p = points[tid];

        for (int j = 0; j < k; j++) {
            Point c = centroids[j];
            double dist = (p.x - c.x) * (p.x - c.x) + (p.y - c.y) * (p.y - c.y) + (p.z - c.z) * (p.z - c.z);

            if (dist < p.minDist) {
                p.minDist = dist;
                p.cluster = j;
            }
        }

        int cluster = p.cluster;
        atomicAdd(&nPoints[cluster], 1);
        atomicAdd(&sumX[cluster], p.x);
        atomicAdd(&sumY[cluster], p.y);
        atomicAdd(&sumZ[cluster], p.z);

        p.minDist = DBL_MAX;
        points[tid] = p;
    }
}

extern "C" void launch_updateCentroids(Point* points, Point* centroids, int* nPoints, double* sumX, double* sumY, double* sumZ, int k, int n) {
    //Dimensions
    int blockSize = 128;
    int numBlocks = (n + blockSize - 1) / blockSize;

    // GPU memory allocation
    hipError_t cudaErr = hipSuccess;
    Point* points_d;
    Point* centroids_d;
    int* nPoints_d;
    double* sumX_d;
    double* sumY_d;
    double* sumZ_d;
    cudaErr = hipMalloc((void **)&points_d, sizeof(Point) * n);
    cudaErr = hipMalloc((void **)&centroids_d, sizeof(Point) * k);
    cudaErr = hipMalloc((void **)&nPoints_d, sizeof(int) * k);
    cudaErr = hipMalloc((void **)&sumX_d, sizeof(double) * k);
    cudaErr = hipMalloc((void **)&sumY_d, sizeof(double) * k);
    cudaErr = hipMalloc((void **)&sumZ_d, sizeof(double) * k);

    if (cudaErr != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory (error code %s)\n", hipGetErrorString(cudaErr));
    }

    // Copy data from CPU to GPU
    cudaErr = hipMemcpy(points_d, points, sizeof(Point) * n, hipMemcpyHostToDevice);
    cudaErr = hipMemcpy(centroids_d, centroids, sizeof(Point) * k, hipMemcpyHostToDevice);
    cudaErr = hipMemcpy(nPoints_d, nPoints, sizeof(int) * k, hipMemcpyHostToDevice);
    cudaErr = hipMemcpy(sumX_d, sumX, sizeof(double) * k, hipMemcpyHostToDevice);
    cudaErr = hipMemcpy(sumY_d, sumY, sizeof(double) * k, hipMemcpyHostToDevice);
    cudaErr = hipMemcpy(sumZ_d, sumZ, sizeof(double) * k, hipMemcpyHostToDevice);

    if (cudaErr != hipSuccess) {
        fprintf(stderr, "Failed to copy data host to device (error code %s)\n", hipGetErrorString(cudaErr));
    }

    updateCentroids<<<numBlocks, blockSize>>> (points_d, centroids_d, nPoints_d, sumX_d, sumY_d, sumZ_d, k, n);

    // Wait for the kernel to finish
    hipDeviceSynchronize();

    // Copy the result back from GPU to CPU
    hipMemcpy(points, points_d, sizeof(Point) * n, hipMemcpyDeviceToHost);
    hipMemcpy(centroids, centroids_d, sizeof(Point) * k, hipMemcpyDeviceToHost);
    hipMemcpy(nPoints, nPoints_d, sizeof(int) * k, hipMemcpyDeviceToHost);
    hipMemcpy(sumX, sumX_d, sizeof(double) * k, hipMemcpyDeviceToHost);
    hipMemcpy(sumY, sumY_d, sizeof(double) * k, hipMemcpyDeviceToHost);
    hipMemcpy(sumZ, sumZ_d, sizeof(double) * k, hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(points_d);
    hipFree(centroids_d);
    hipFree(nPoints_d);
    hipFree(sumX_d);
    hipFree(sumY_d);
    hipFree(sumZ_d);

}


